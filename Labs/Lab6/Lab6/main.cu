
#include <hip/hip_runtime.h>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void VecAdd(const float *A, const float *B, float* C, long unsigned int n) {
	long unsigned int ID = blockIdx.x * blockDim.x + threadIdx.x;

    if (ID < n)
        C[ID] = A[ID] + B[ID];
}

int main (int argc, char *argv[]){
    float *A_h, *B_h, *C_h;
    float *A_d, *B_d, *C_d;
	int blockSize, gridSize;
	long unsigned int VecSize, Limit = (1 << 25); // Limit 4MB

	FILE *fp = fopen("results.csv", "w");
	fprintf(fp,"Method, Size, Time\n");

	printf("\nCudaMemcpy Test\n");
	for(VecSize = 8; VecSize <= Limit; VecSize *= 2){
		clock_t start = clock(); //Starts
		A_h = (float*) malloc( sizeof(float) * VecSize );
		B_h = (float*) malloc( sizeof(float) * VecSize );
		C_h = (float*) malloc( sizeof(float) * VecSize );
		
		for (long unsigned int i=0; i < VecSize; i++) {
			A_h[i] = 1.0f;
			B_h[i] = 2.0f;
		}

		hipDeviceSynchronize();

		hipMalloc(&A_d, sizeof(float) * VecSize);
		hipMalloc(&B_d, sizeof(float) * VecSize);
		hipMalloc(&C_d, sizeof(float) * VecSize);

		hipMemcpy(A_d, A_h, sizeof(float) * VecSize, hipMemcpyHostToDevice);
		hipMemcpy(B_d, B_h, sizeof(float) * VecSize, hipMemcpyHostToDevice);
		
		hipDeviceSynchronize();

		blockSize = 32;
		gridSize = (int)ceil((float)VecSize/blockSize);
		
		VecAdd<<<gridSize, blockSize>>>(A_d, B_d, C_d, VecSize);

		hipMemcpy(C_h, C_d, sizeof(float) * VecSize, hipMemcpyDeviceToHost);

		hipDeviceSynchronize();

		free(A_h);
		free(B_h);
		free(C_h);

		hipFree(A_d);
		hipFree(B_d);
		hipFree(C_d);

		hipDeviceSynchronize();
		clock_t end = clock(); //Ends

		//Measure Data
		double elasped_secs = (((double) end - (double) start) / CLOCKS_PER_SEC) * 1000000;
		printf("Size: %ld, Time: %f us\n", sizeof(float) * VecSize, elasped_secs);
		fprintf(fp,"%s, %ld, %f\n", "CudaMemcpy", sizeof(float) * VecSize, elasped_secs);
	}

	printf("\nPinned Memory(cudaHostAlloc) Test\n");
	for(VecSize = 8; VecSize <= Limit; VecSize *= 2){
		clock_t start = clock(); //Starts
		hipDeviceSynchronize();

		hipHostAlloc(&A_h, sizeof(float) * VecSize, hipHostMallocDefault);
		hipHostAlloc(&B_h, sizeof(float) * VecSize, hipHostMallocDefault);
		hipHostAlloc(&C_h, sizeof(float) * VecSize, hipHostMallocDefault);
		
		for (long unsigned int i=0; i < VecSize; i++) {
			A_h[i] = 1.0f;
			B_h[i] = 2.0f;
		}
		
		hipHostGetDevicePointer(&A_d, A_h, 0);
		hipHostGetDevicePointer(&B_d, B_h, 0);
		hipHostGetDevicePointer(&C_d, C_h, 0);
		
		hipDeviceSynchronize();

		blockSize = 32;
		gridSize = (int)ceil((float)VecSize/blockSize);
		
		VecAdd<<<gridSize, blockSize>>>(A_d, B_d, C_d, VecSize);

		hipDeviceSynchronize();

		hipHostFree(A_h);
		hipHostFree(B_h);
		hipHostFree(C_h);

		A_d = NULL;
		B_d = NULL;
		C_d = NULL;

		hipDeviceSynchronize();
		clock_t end = clock(); //Ends

		//Measure Data
		double elasped_secs = (((double) end - (double) start) / CLOCKS_PER_SEC) * 1000000;
		printf("Size: %ld, Time: %f us\n", sizeof(float) * VecSize, elasped_secs);
		fprintf(fp,"%s, %ld, %f\n", "hipHostAlloc", sizeof(float) * VecSize, elasped_secs);
	}

	printf("\nUnified Virtual Memory(cudaMallocManaged) Test\n");
	for(VecSize = 8; VecSize <= Limit; VecSize *= 2){
		clock_t start = clock(); //Starts
		hipDeviceSynchronize();

		hipMallocManaged(&A_h, sizeof(float) * VecSize);
		hipMallocManaged(&B_h, sizeof(float) * VecSize);
		hipMallocManaged(&C_h, sizeof(float) * VecSize);
		
		for (long unsigned int i=0; i < VecSize; i++) {
			A_h[i] = 1.0f;
			B_h[i] = 2.0f;
		}
		
		hipDeviceSynchronize();

		blockSize = 32;
		gridSize = (int)ceil((float)VecSize/blockSize);
		
		VecAdd<<<gridSize, blockSize>>>(A_h, B_h, C_h, VecSize);

		hipDeviceSynchronize();

		hipFree(A_h);
		hipFree(B_h);
		hipFree(C_h);

		hipDeviceSynchronize();
		clock_t end = clock(); //Ends

		//Measure Data
		double elasped_secs = (((double) end - (double) start) / CLOCKS_PER_SEC) * 1000000;
		printf("Size: %ld, Time: %f us\n", sizeof(float) * VecSize, elasped_secs);
		fprintf(fp,"%s, %ld, %f\n", "hipMallocManaged", sizeof(float) * VecSize, elasped_secs);
	}

	fclose(fp);

    return 0;
}
